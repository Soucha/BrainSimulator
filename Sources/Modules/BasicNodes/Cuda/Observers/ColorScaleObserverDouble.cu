#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <hip/hip_runtime_api.h>
#include <> 
#include <> 
#include <builtin_types.h> 
#include <hip/hip_vector_types.h> 
#include <vector_functions.h> 
#include <float.h>

#include "ColorHelpers.cu"

extern "C"  
{	
	//kernel code
	__global__ void ColorScaleObserverDouble(double* values, int method, int scale, float minValue, float maxValue, unsigned int* pixels, int numOfPixels)
	{		
		int id = blockDim.x*blockIdx.y*gridDim.x	
			+ blockDim.x*blockIdx.x				
			+ threadIdx.x;

		if(id < numOfPixels) //id of the thread is valid
		{	
			pixels[id] = float_to_uint_rgba(values[id], method, scale, minValue, maxValue);
		}
	}


	__global__ void DrawVectorsKernel(float* values, int elements, float maxValue, unsigned int* pixels, int numOfPixels) 
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		if(id < numOfPixels) //id of the thread is valid
		{
			float x = values[id] / maxValue;
			float y = values[numOfPixels + id] / maxValue;

			if (elements == 2) {

				float hue = atan2f(x, y) / HIP_PI_F * 0.5f + 0.5f;				
				float value = fminf(sqrtf(x * x + y * y), 1.0f);

				pixels[id] = hsva_to_uint_rgba(hue, 1.0f, value, 1.0f);
			}
			else {
				
				float z = values[2 * numOfPixels + id] / maxValue;

				x = fminf(fmaxf(x, -1), 1);
				y = fminf(fmaxf(y, -1), 1);
				z = fminf(fmaxf(z, -1), 1);

				unsigned char red = (unsigned char) __float2uint_rn(127.5f * (x + 1));
				unsigned char green = (unsigned char) __float2uint_rn(127.5f * (y + 1));
				unsigned char blue = (unsigned char) __float2uint_rn(127.5f * (z + 1));		

				pixels[id] = (0xFF << 24) | (red << 16) | (green << 8) | blue;		
			}
		}
	}
}