#include "hip/hip_runtime.h"
//#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C"
{
	__global__ void ShuffleRGB(float* input, float* output, int size)
	{
		int id = blockDim.x * blockIdx.y * gridDim.x
			+ blockDim.x * blockIdx.x
			+ threadIdx.x;

		if (id < size)
		{
			//int index = id / 3 + (id % 3) * (size / 3);
			output[id / 3 + (id % 3) * (size / 3)] = input[id];
		}
	}
}